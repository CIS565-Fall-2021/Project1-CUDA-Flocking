#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"
#include "cVec.h"


#define GRID_LOOP 0
#define ALWAYS_27 0
/* always checking 27 cells only enabled if GRID_LOOP is 0*/

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/* Check for CUDA errors; print and exit if there was a problem */
void checkCUDAError(const char *msg, int line = -1)
{
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		if (line >= 0) {
			fprintf(stderr, "Line %d: ", line);
		}
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

/* better max and min functions*/
template <typename T>
__host__ __device__ T max(T v) {
	return v;
}

template <typename T, typename... U>
__host__ __device__ T max(T v1, T v2, U ... vs) {
	return max(v1 > v2 ? v1 : v2, vs...);
}

template <typename T>
__host__ __device__ T min(T v) {
	return v;
}

template <typename T, typename... U>
__host__ __device__ T min(T v1, T v2, U ... vs) {
	return max(v1 < v2 ? v1 : v2, vs...);
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define block_size 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1_dist 5.0f
#define rule2_dist 3.0f
#define rule3_dist 5.0f

#define rule1_scale 0.01f
#define rule2_scale 0.1f
#define rule3_scale 0.1f

#define max_speed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int num_boids;
dim3 threadsPerBlock(block_size);


using glm::vec3;

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
cu::cPtr<vec3> dv_pos;
cu::cPtr<vec3> dv_vel1;
cu::cPtr<vec3> dv_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
cu::cPtr<int> dv_particle_grid_indices; // What grid cell is this particle in? (grid cell index)
cu::cPtr<int> dv_particle_array_indices; // What index in dv_pos and dev_velX represents this particle? (boid index)

thrust::device_ptr<int> dv_thrust_particle_array_indices;
thrust::device_ptr<int> dv_thrust_particle_grid_indices;

cu::cPtr<int> dv_gridcell_start_indices; // What part of dev_particleArrayIndices belongs
cu::cPtr<int> dv_gridcell_end_indices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells:
cu::cPtr<vec3> dv_pos2; /* we already have two dv_vel so we can use the second as the rearranged version */

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a)
{
	a = (a + 0x7ed55d16) + (a << 12);
	a = (a ^ 0xc761c23c) ^ (a >> 19);
	a = (a + 0x165667b1) + (a << 5);
	a = (a + 0xd3a2646c) ^ (a << 9);
	a = (a + 0xfd7046c5) + (a << 3);
	a = (a ^ 0xb55a4f09) ^ (a >> 16);
	return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ vec3 generateRandomVec3(float time, int index)
{
	thrust::default_random_engine rng(hash((int)(index * time)));
	thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

	return vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}


/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, vec3 *arr, float scale)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N) {
		vec3 rand = generateRandomVec3(time, index);
		arr[index] = scale * rand;
	}
}

/* Initialize memory, update some globals */
void Boids::initSimulation(int N)
{
	num_boids = N;
	dim3 fullBlocksPerGrid((N + block_size - 1) / block_size);

	// LOOK-1.2 - This is basic CUDA memory management and error checking.
	dv_pos = cu::make<vec3>(N);
	dv_vel1 = cu::make<vec3>(N);
	dv_vel2 = cu::make<vec3>(N);

	// LOOK-1.2 - This is a typical CUDA kernel invocation.
	kernGenerateRandomPosArray<<<fullBlocksPerGrid, block_size>>>(1, num_boids, dv_pos.get(), scene_scale);
	checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

	// LOOK-2.1 computing grid params
	gridCellWidth = 2.0f * max(rule1_dist, rule2_dist, rule3_dist); //std::max(std::max(rule1_dist, rule2_dist), rule3_dist);
	int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
	gridSideCount = 2 * halfSideCount;

	gridCellCount = gridSideCount * gridSideCount * gridSideCount;
	gridInverseCellWidth = 1.0f / gridCellWidth;
	float halfGridWidth = gridCellWidth * halfSideCount;
	gridMinimum.x -= halfGridWidth;
	gridMinimum.y -= halfGridWidth;
	gridMinimum.z -= halfGridWidth;

	// TODO-2.1 TODO-2.3 - Allocate additional buffers here.
	
	dv_particle_grid_indices = cu::make<int>(N);
	dv_particle_array_indices = cu::make<int>(N);

	dv_gridcell_start_indices = cu::make<int>(gridCellCount);
	dv_gridcell_end_indices = cu::make<int>(gridCellCount);

	dv_thrust_particle_grid_indices = thrust::device_ptr<int>(dv_particle_grid_indices.get());
	dv_thrust_particle_array_indices = thrust::device_ptr<int>(dv_particle_array_indices.get());

	dv_pos2 = cu::make<vec3>(N);

	hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/* Copy the boid positions into the VBO so that they can be drawn by OpenGL */
__global__ void kernCopyPositionsToVBO(int N, const vec3 *pos, float *vbo, float s_scale)
{
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	float c_scale = -1.0f / s_scale;

	if (index < N) {
		vbo[4 * index + 0] = pos[index].x * c_scale;
		vbo[4 * index + 1] = pos[index].y * c_scale;
		vbo[4 * index + 2] = pos[index].z * c_scale;
		vbo[4 * index + 3] = 1.0f;
	}
}

__global__ void kernCopyVelocitiesToVBO(int N, const vec3 *vel, float *vbo, float s_scale)
{
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if (index < N) {
		vbo[4 * index + 0] = vel[index].x + 0.3f;
		vbo[4 * index + 1] = vel[index].y + 0.3f;
		vbo[4 * index + 2] = vel[index].z + 0.3f;
		vbo[4 * index + 3] = 1.0f;
	}
}

/* Wrapper for call to the kernCopyboidsToVBO CUDA kernel */
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities)
{
	dim3 fullBlocksPerGrid((num_boids + block_size - 1) / block_size);

	kernCopyPositionsToVBO <<<fullBlocksPerGrid, block_size>>>(num_boids, dv_pos.get(), vbodptr_positions, scene_scale);
	kernCopyVelocitiesToVBO <<<fullBlocksPerGrid, block_size>>>(num_boids, dv_vel1.get(), vbodptr_velocities, scene_scale);

	checkCUDAErrorWithLine("copyBoidsToVBO failed!");

	hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/* p is position of this boid, boid_pos, boid_vel are of neighbour we*/
__device__ __forceinline__ void apply_rules(const vec3 &p, const vec3 &boid_pos, const vec3 &boid_vel,
	vec3 *perceived_center, vec3 *perceived_vel, vec3 *c,
	int *neighbour_count_p, int *neighbour_count_v)
{
	float len = glm::distance(boid_pos, p);

	// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
	if (len < rule1_dist)  {
		*perceived_center += boid_pos;
		(*neighbour_count_p)++;
	}

	// Rule 2: boids try to stay a distance d away from each other
	if (len < rule2_dist)
		*c -= (boid_pos - p);

	// Rule 3: boids try to match the speed of surrounding boids
	if (len < rule3_dist) {
		*perceived_vel += boid_vel;
		(*neighbour_count_v)++;
	}
}

__device__ __forceinline__ vec3 out_vel(const vec3 &p, vec3 v, const vec3 &perceived_center, const vec3 &perceived_vel,
	const vec3 &c, int neighbour_count_p, int neighbour_count_v)
{
	if (neighbour_count_p > 0)
		v += (perceived_center / (float) neighbour_count_p - p) * rule1_scale;
	v += c * rule2_scale;
	if (neighbour_count_v > 0)
		v += perceived_vel / (float) neighbour_count_v * rule3_scale;

	return v * max_speed / max(max_speed, glm::length(v)); /* clamp to max_speed */
}


/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
* Compute the new velocity on the body with index `idx` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__global__ void kern_update_vel_brute_force(int N, const vec3 *pos, const vec3 *vel1, vec3 *vel2)
{
	int idx= threadIdx.x + (blockIdx.x * blockDim.x);

	vec3 v = vel1[idx];
	vec3 p = pos[idx];

	// Compute a new velocity based on pos and vel1

	vec3 perceived_center(0.0f);
	vec3 perceived_vel(0.0f);
	int neighbour_count_p = 0, neighbour_count_v = 0;
	vec3 c(0.0f);

	for (int i = 0; i < N; i++) {
		if (i != idx)
			apply_rules(p, pos[i], vel1[i], &perceived_center, &perceived_vel, &c, &neighbour_count_p, &neighbour_count_v);
	}

	vel2[idx] = out_vel(p, v, perceived_center, perceived_vel, c, neighbour_count_p,  neighbour_count_v);
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, vec3 *pos, const vec3 *vel)
{
	// Update position by velocity
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N)
		return;

	vec3 thisPos = pos[index] + vel[index] * dt;

	// Wrap the boids around so we don't lose them
	thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x > scene_scale ? -scene_scale : thisPos.x;
	thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y > scene_scale ? -scene_scale : thisPos.y;
	thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z > scene_scale ? -scene_scale : thisPos.z;

	pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution)
{
	return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
	vec3 gridMin, float inverseCellWidth,
	const vec3 *pos, int * __restrict__ indices, int * __restrict__ gridIndices) {
		// TODO-2.1
		// - Label each boid with the index of its grid cell.
		// - Set up a parallel array of integer indices as pointers to the actual
		//   boid data in pos and vel1/vel2
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N)
		return;
	
	vec3 offset = (pos[index] - gridMin) * inverseCellWidth;
	gridIndices[index] = gridIndex3Dto1D(offset.x, offset.y, offset.z, gridResolution);
	indices[index] = index;
}


__global__ void kernIdentifyCellStartEnd(int N, const int *particleGridIndices,
	int *__restrict__ gridCellStartIndices, int *__restrict__ gridCellEndIndices) {
	// TODO-2.1
	// Identify the start point of each cell in the gridIndices array.
	// This is basically a parallel unrolling of a loop that goes
	// "this index doesn't match the one before it, must be a new cell!"
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N)
		return;

	if (index == 0)
		gridCellStartIndices[particleGridIndices[0]] = 0;
	else if (particleGridIndices[index] != particleGridIndices[index-1])
		gridCellStartIndices[particleGridIndices[index]] = index;
	if (index == N-1)
		gridCellEndIndices[particleGridIndices[N-1]] = N-1;
	else if (particleGridIndices[index] != particleGridIndices[index+1])
		gridCellEndIndices[particleGridIndices[index]] = index;

}

/* boid_search_apply iterates over the boids of the neighbouring cells and executes function apply_rules_b2 with
 * their indices: in the case of the scattered grid, this is the indices that go into particleGridIndices
 * in the case of coherent grid, the indices are directly the indices of the boids in the pos and vel arrays */
/* F is a function that takes in the neighbour boid index/particleArrayIndex index as determined by this function */
template <typename F>
__device__ __forceinline__ void boid_search_apply(const vec3 &p, int gridResolution, vec3 gridMin, float inverseCellWidth,
	const int *gridCellStartIndices, const int *gridCellEndIndices, F apply_rules_b2)
{
	// - Identify the grid cell that this particle is in
	// - Identify which cells may contain neighbors. This isn't always 8.

#if GRID_LOOP
	float dist = max(rule1_dist, rule2_dist, rule3_dist) * inverseCellWidth;
	vec3 minv = (p - gridMin) * inverseCellWidth - dist; /* grid looping optimization */
	vec3 maxv = (p - gridMin) * inverseCellWidth + dist;
#else
 #if ALWAYS_27
	vec3 minv = (p - gridMin) * inverseCellWidth - 1.0f; /* always checks 27 squares */
	vec3 maxv = (p - gridMin) * inverseCellWidth + 1.0f;

 #else
	vec3 minv = (p - gridMin) * inverseCellWidth - 0.5f; /* always checks 8 squares */
	vec3 maxv = (p - gridMin) * inverseCellWidth + 0.5f;
 #endif
#endif

	dim3 mincoords = dim3(max(0, (int) minv.x), max(0, (int) minv.y), max(0, (int) minv.z));
	dim3 maxcoords = dim3(min(gridResolution - 1, (int) maxv.x), min(gridResolution - 1, (int) maxv.y), min(gridResolution - 1, (int) maxv.z));

	for (int z = mincoords.z; z <= maxcoords.z; z++) {
		for (int y = mincoords.y; y <= maxcoords.y; y++) {
			for (int x = mincoords.x; x <= maxcoords.x; x++) {
				// - For each cell, read the start/end indices in the boid pointer array.
				int start = gridCellStartIndices[gridIndex3Dto1D(x, y, z, gridResolution)];
				int end = gridCellEndIndices[gridIndex3Dto1D(x, y, z, gridResolution)];
				if (start == -1)
					continue;

				// - Access each boid in the cell and compute velocity change from
				//   the boids rules, if this boid is within the neighborhood distance.
				for (int i = start; i <= end; i++) {
					apply_rules_b2(i);
				}
			}
		}
	}

}

__global__ void kernUpdateVelNeighborSearchScattered(
	int N, int gridResolution, vec3 gridMin,
	float inverseCellWidth,
	const int *gridCellStartIndices, const int *gridCellEndIndices,
	const int *particleArrayIndices,
	const vec3 *pos, const vec3 *vel1, vec3 *vel2) {
	// TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
	// the number of boids that need to be checked.

	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (idx >= N)
		return;

	vec3 p = pos[idx];
	vec3 v = vel1[idx];

	vec3 perceived_center(0.0f);
	vec3 perceived_vel(0.0f);
	int neighbour_count_p = 0, neighbour_count_v = 0;
	vec3 c(0.0f);


	boid_search_apply(p, gridResolution, gridMin, inverseCellWidth, gridCellStartIndices, gridCellEndIndices,
		[&] (int i) {
			int b2 = particleArrayIndices[i];
			if (idx != b2)
				apply_rules(p, pos[b2], vel1[b2], &perceived_center, &perceived_vel, &c, &neighbour_count_p, &neighbour_count_v);
		});


	vel2[idx] = out_vel(p, v, perceived_center, perceived_vel, c, neighbour_count_p,  neighbour_count_v);
}

__global__ void kernUpdateVelNeighborSearchCoherent(
	int N, int gridResolution, vec3 gridMin,
	float inverseCellWidth,
	const int *gridCellStartIndices, const int *gridCellEndIndices,
	const vec3 *pos, const vec3 *vel1, vec3 *vel2) {

	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (idx >= N)
		return;

	vec3 p = pos[idx];
	vec3 v = vel1[idx];

	vec3 perceived_center(0.0f);
	vec3 perceived_vel(0.0f);
	int neighbour_count_p = 0, neighbour_count_v = 0;
	vec3 c(0.0f);

	boid_search_apply(p, gridResolution, gridMin, inverseCellWidth, gridCellStartIndices, gridCellEndIndices,
		[&] (int b2) {
			if (idx != b2)
				apply_rules(p, pos[b2], vel1[b2], &perceived_center, &perceived_vel, &c, &neighbour_count_p, &neighbour_count_v);
		});

	vel2[idx] = out_vel(p, v, perceived_center, perceived_vel, c, neighbour_count_p,  neighbour_count_v);
}


/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt)
{
	// TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
	// TODO-1.2 ping-pong the velocity buffers
	
	int blocks_per_grid = (num_boids + block_size - 1) / block_size;

	kernUpdatePos<<<blocks_per_grid, block_size>>>(num_boids, dt, dv_pos.get(), dv_vel1.get());
	checkCUDAErrorWithLine("kernUpdatePos failed!");

	kern_update_vel_brute_force<<<blocks_per_grid, block_size>>>(num_boids, dv_pos.get(), dv_vel1.get(), dv_vel2.get());
	checkCUDAErrorWithLine("kern_update_vel_brute_force failed!");

	std::swap(dv_vel1, dv_vel2);
}

void Boids::stepSimulationScatteredGrid(float dt)
{
	int blocks_per_grid = (num_boids + block_size - 1) / block_size;
	
	// - label each particle with its array index as well as its grid index.
	kernComputeIndices<<<blocks_per_grid, block_size>>>(num_boids, gridSideCount, gridMinimum,
		gridInverseCellWidth, dv_pos.get(), dv_particle_array_indices.get(), dv_particle_grid_indices.get());
	checkCUDAErrorWithLine("kernComputeIndices failed!");

	// - Unstable key sort using Thrust
	thrust::sort_by_key(dv_thrust_particle_grid_indices, dv_thrust_particle_grid_indices + num_boids, dv_thrust_particle_array_indices);

	// - Naively unroll the loop for finding the start and end indices of each
	//   cell's data pointers in the array of boid indices
	cu::set(dv_gridcell_start_indices, -1, gridCellCount);

	kernIdentifyCellStartEnd<<<blocks_per_grid, block_size>>>(num_boids, dv_particle_grid_indices.get(),
		dv_gridcell_start_indices.get(), dv_gridcell_end_indices.get());
	checkCUDAErrorWithLine("kernIdentifyCellStartEnd failed!");

	// - Perform velocity updates using neighbor search
	kernUpdateVelNeighborSearchScattered<<<blocks_per_grid, block_size>>>(num_boids, gridSideCount, gridMinimum, gridInverseCellWidth,
		dv_gridcell_start_indices.get(), dv_gridcell_end_indices.get(), dv_particle_array_indices.get(),
		dv_pos.get(), dv_vel1.get(), dv_vel2.get());
	checkCUDAErrorWithLine("kernUpdateVelNeighborSearchScattered failed!");

	// - Update positions
	kernUpdatePos<<<blocks_per_grid, block_size>>>(num_boids, dt, dv_pos.get(), dv_vel2.get());
	checkCUDAErrorWithLine("kernUpdatePos failed!");
	
	// - Ping-pong buffers
	std::swap(dv_vel1, dv_vel2);
}


__global__ void kern_rearrange_boid_data(int N, const int* indices, const vec3* pos, const vec3* vel,
	vec3* __restrict__ pos2, vec3* __restrict__ vel2)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index >= N)
		return;

	pos2[index] = pos[indices[index]];
	vel2[index] = vel[indices[index]];
}

void Boids::stepSimulationCoherentGrid(float dt)
{
	// TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
	// Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
	// In Parallel:

	int blocks_per_grid = (num_boids + block_size - 1) / block_size;

	// - Label each particle with its array index as well as its grid index.
	kernComputeIndices<<<blocks_per_grid, block_size>>>(num_boids, gridSideCount, gridMinimum,
		gridInverseCellWidth, dv_pos.get(), dv_particle_array_indices.get(), dv_particle_grid_indices.get());
	checkCUDAErrorWithLine("kernComputeIndices failed!");

	// - Unstable key sort using Thrust
	thrust::sort_by_key(dv_thrust_particle_grid_indices, dv_thrust_particle_grid_indices + num_boids, dv_thrust_particle_array_indices);

	// - Naively unroll the loop for finding the start and end indices of each
	//   cell's data pointers in the array of boid indices
	cu::set(dv_gridcell_start_indices, -1, gridCellCount);

	kernIdentifyCellStartEnd<<<blocks_per_grid, block_size>>>(num_boids, dv_particle_grid_indices.get(),
		dv_gridcell_start_indices.get(), dv_gridcell_end_indices.get());
	checkCUDAErrorWithLine("kernIdentifyCellStartEnd failed!");

	// - use the rearranged array index buffer to reshuffle all the particle data in the simulation array.
	kern_rearrange_boid_data<<<blocks_per_grid, block_size>>>(num_boids, dv_particle_array_indices.get(), dv_pos.get(), dv_vel1.get(), dv_pos2.get(), dv_vel2.get());
	checkCUDAErrorWithLine("kern_rearrange_boid_data failed!");

	// - Perform velocity updates using neighbor search
	kernUpdateVelNeighborSearchCoherent<<<blocks_per_grid, block_size>>>(num_boids, gridSideCount, gridMinimum, gridInverseCellWidth,
		dv_gridcell_start_indices.get(), dv_gridcell_end_indices.get(), dv_pos2.get(), dv_vel2.get(), dv_vel1.get());
	checkCUDAErrorWithLine("kernUpdateVelNeighborSearchCoherent failed!");

	// - Update positions
	kernUpdatePos<<<blocks_per_grid, block_size>>>(num_boids, dt, dv_pos2.get(), dv_vel1.get());
	checkCUDAErrorWithLine("kernUpdatePos failed!");

	// - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
	std::swap(dv_pos, dv_pos2); /* vel1 contains the final velocities so only need to update pos */
	
}

void Boids::endSimulation()
{
	cu::del(dv_pos);
	cu::del(dv_vel1);
	cu::del(dv_vel2);

	// TODO-2.1 TODO-2.3 - Free any additional buffers here.
	cu::del(dv_particle_grid_indices);
	cu::del(dv_particle_array_indices);
	cu::del(dv_gridcell_start_indices);
	cu::del(dv_gridcell_end_indices);

	cu::del(dv_pos2);
}

void Boids::unitTest()
{
	// LOOK-1.2 Feel free to write additional tests here.

	// test unstable sort
	int N = 10;
	cu::cVec<int> dev_intKeys(N);
	cu::cVec<int> dev_intValues(N);

	std::unique_ptr<int[]>intKeys{ new int[N] };
	std::unique_ptr<int[]>intValues{ new int[N] };

	intKeys[0] = 0; intValues[0] = 0;
	intKeys[1] = 1; intValues[1] = 1;
	intKeys[2] = 0; intValues[2] = 2;
	intKeys[3] = 3; intValues[3] = 3;
	intKeys[4] = 0; intValues[4] = 4;
	intKeys[5] = 2; intValues[5] = 5;
	intKeys[6] = 2; intValues[6] = 6;
	intKeys[7] = 0; intValues[7] = 7;
	intKeys[8] = 5; intValues[8] = 8;
	intKeys[9] = 6; intValues[9] = 9;


	dim3 fullBlocksPerGrid((N + block_size - 1) / block_size);

	std::cout << "before unstable sort: " << std::endl;
	for (int i = 0; i < N; i++) {
		std::cout << "  key: " << intKeys[i];
		std::cout << " value: " << intValues[i] << std::endl;
	}

	// How to copy data to the GPU
	cu::copy(dev_intKeys.ptr(), intKeys.get(), N);
	cu::copy(dev_intValues.ptr(), intValues.get(), N);

	// Wrap device vectors in thrust iterators for use with thrust.
	thrust::device_ptr<int> dev_thrust_keys(dev_intKeys.get());
	thrust::device_ptr<int> dev_thrust_values(dev_intValues.get());
	// LOOK-2.1 Example for using thrust::sort_by_key
	thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

	// How to copy data back to the CPU side from the GPU
	cu::copy(intKeys.get(), dev_intKeys.ptr(), N);
	cu::copy(intValues.get(), dev_intValues.ptr(), N);

	std::cout << "after unstable sort: " << std::endl;
	for (int i = 0; i < N; i++) {
		std::cout << "  key: " << intKeys[i];
		std::cout << " value: " << intValues[i] << std::endl;
	}

	return;
}
